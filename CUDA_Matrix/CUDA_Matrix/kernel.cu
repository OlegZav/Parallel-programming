#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <windows.h>

using namespace std;

__global__ void matrixMult(const int* A, const int* B, int* C, int matrix)
{
    int i = matrix * (blockDim.y * blockIdx.y + threadIdx.y);
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int sum = 0;

    for (int k = 0; k < matrix; k++)
        sum += A[i + k] * B[k * matrix + j];

    int ind = matrix * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    C[ind] = sum;
}

void error_to_console(string out_string, string out_retry)
{
    cin.clear();
    cin.ignore(cin.rdbuf()->in_avail());
    cout << out_string << endl;
    cout << out_retry;
}

int main(int argc, char** argv) {

    SetConsoleOutputCP(CP_UTF8);
    
    int threads, matrix;
    bool flag_thread = true, flag_matrix = true;
    string thread_input, matrix_size;
    
    cout << "Программа вычисляет произведение матриц при помощи библиотеки CUDA" << endl;
    cout << "Введите неоходимое количество потоков = ";

    while (flag_thread == true)
    {
        try
        {
            cin >> thread_input;
            threads = stoi(thread_input);

            if (threads < 0)
                error_to_console("Введено число меньшее 1. Введите значение повторно",
                    "Количество потоков = ");
            else
                flag_thread = false;
        }
        catch (invalid_argument)
        {
            error_to_console("Введён некорректный элемент. Введите значение повторно",
                "Количество потоков = ");
        }
    }

    cout << "Введите размер матрицы = ";

    while (flag_matrix == true)
    {
        try
        {
            cin >> matrix_size;
            matrix = stoi(matrix_size);

            if (matrix < 0)
                error_to_console("Введено число меньшее 1. Введите значение повторно",
                    "Введите размер матрицы = ");
            else
                flag_matrix = false;
        }
        catch (invalid_argument)
        {
            error_to_console("Введён некорректный элемент. Введите значение повторно",
                "Введите размер матрицы = ");
        }
    }

    size_t size = matrix * matrix * sizeof(int);

    int* a = (int*)malloc(size);
    int* b = (int*)malloc(size);
    int* c = (int*)malloc(size);

    for (int i = 0; i < matrix; i++) {
        for (int j = 0; j < matrix; j++) {
            a[i * matrix + j] = i * j;
            b[i * matrix + j] = i * j;
        }
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int* calcA = NULL;
    hipMalloc((void**)&calcA, size);

    int* calcB = NULL;
    hipMalloc((void**)&calcB, size);

    int* calcC = NULL;
    hipMalloc((void**)&calcC, size);

    hipMemcpy(calcA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(calcB, b, size, hipMemcpyHostToDevice);

    cout << "Программа начала свою работу" << endl;
    
    dim3 threadsPerBlock = dim3(threads, threads);
    dim3 blocksPerGrid = dim3(matrix / threads, matrix / threads);

    hipEventRecord(start, 0);
    matrixMult <<< blocksPerGrid, threadsPerBlock >>> (calcA, calcB, calcC, matrix);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);
    double time = kernelTime;
    cout << "Потоков на блок: " << threads
        << "; блоки в сетке: " << matrix / threads
        << "; время работы: " << kernelTime / 1000  << " секунд" << endl;
    hipMemcpy(c, calcC, size, hipMemcpyDeviceToHost);
    
    hipFree(calcA);
    hipFree(calcB);
    hipFree(calcC);
    free(a);
    free(b);
    free(c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
